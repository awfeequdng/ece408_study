#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vecAddKernel(int *d_a, int *d_b, int *d_c, int n) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) {
		d_c[id] = d_a[id] + d_b[id];
	}
}

int main() {

	const int MAX = 500;
	int a[MAX] = {1, 2,3, 4, 5};
	int b[MAX] = {2,3, 4, 5, 6};
	int c[MAX] = {0};
	
	int *d_a;
	int *d_b;
	int *d_c;
	hipMalloc(&d_a, sizeof(a));
	hipMalloc(&d_b, sizeof(a));
	hipMalloc(&d_c, sizeof(a));

	hipMemcpy(d_a, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(a), hipMemcpyHostToDevice);
	int N = 1020;
	dim3 grid(ceil(N/256));
	dim3 block(256);
		
	vecAddKernel<<<grid, block>>>(d_a, d_b, d_c, MAX);
	//cudaDeviceSynchronize();
	hipMemcpy(c, d_c, sizeof(a), hipMemcpyDeviceToHost);
	for(int i = 0; i < MAX; i++) {
		std::cout << " " << c[i];
		if (i % 10 == 0) std::cout << std::endl;
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}


